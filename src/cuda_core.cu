#include "hip/hip_runtime.h"
/* XMRig
* Copyright 2010      Jeff Garzik <jgarzik@pobox.com>
* Copyright 2012-2014 pooler      <pooler@litecoinpool.org>
* Copyright 2014      Lucas Jones <https://github.com/lucasjones>
* Copyright 2014-2016 Wolf9466    <https://github.com/OhGodAPet>
* Copyright 2016      Jay D Dee   <jayddee246@gmail.com>
* Copyright 2017-2018 XMR-Stak    <https://github.com/fireice-uk>, <https://github.com/psychocrypt>
* Copyright 2018      Lee Clagett <https://github.com/vtnerd>
* Copyright 2016-2018 XMRig       <https://github.com/xmrig>, <support@xmrig.com>
*
*   This program is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, either version 3 of the License, or
*   (at your option) any later version.
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License
*   along with this program. If not, see <http://www.gnu.org/licenses/>.
*/


#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "CudaCryptonightR_gen.h"
#include "common/utils/timestamp.h"
#include "crypto/cn/CnAlgo.h"

#ifdef _WIN32
#include <Windows.h>
static void compat_usleep(int waitTime)
{
    if (waitTime > 0) {
        if (waitTime > 100) {
            // use a waitable timer for larger intervals > 0.1ms

            HANDLE timer;
            LARGE_INTEGER ft;

            ft.QuadPart = -10ll * int64_t(waitTime); // Convert to 100 nanosecond interval, negative value indicates relative time

            timer = CreateWaitableTimer(nullptr, TRUE, nullptr);
            SetWaitableTimer(timer, &ft, 0, nullptr, nullptr, 0);
            WaitForSingleObject(timer, INFINITE);
            CloseHandle(timer);
        } else {
            // use a polling loop for short intervals <= 0.1ms

            LARGE_INTEGER perfCnt, start, now;
            int64_t elapsed;

            QueryPerformanceFrequency(&perfCnt);
            QueryPerformanceCounter(&start);
            do {
                SwitchToThread();
                QueryPerformanceCounter(&now);
                elapsed = static_cast<int64_t>(((now.QuadPart - start.QuadPart) / static_cast<float>(perfCnt.QuadPart) * 1000 * 1000));
            } while (elapsed < static_cast<int64_t>(waitTime));
        }
    }
}
#else
#include <unistd.h>
static inline void compat_usleep(int waitTime)
{
    usleep(static_cast<uint64_t>(waitTime));
}
#endif

#include "cryptonight.h"
#include "cuda_extra.h"
#include "cuda_aes.hpp"
#include "cuda_device.hpp"
#include "cuda_fast_int_math_v2.hpp"
#include "cuda_fast_div_heavy.hpp"

#if defined(__x86_64__) || defined(_M_AMD64) || defined(__LP64__)
#   define _ASM_PTR_ "l"
#else
#   define _ASM_PTR_ "r"
#endif

/* sm_2X is limited to 2GB due to the small TLB
 * therefore we never use 64bit indices
 */
#if defined(XMRIG_LARGEGRID) && (__CUDA_ARCH__ >= 300)
typedef uint64_t IndexType;
#else
typedef int IndexType;
#endif

template< typename T >
__device__ __forceinline__ T loadGlobal64( T * const addr )
{
#   if (__CUDA_ARCH__ < 700)
    T x;
    asm volatile( "ld.global.cg.u64 %0, [%1];" : "=l"( x ) : _ASM_PTR_(addr));
    return x;
#   else
    return *addr;
#   endif
}

template< typename T >
__device__ __forceinline__ T loadGlobal32( T * const addr )
{
#   if (__CUDA_ARCH__ < 700)
    T x;
    asm volatile( "ld.global.cg.u32 %0, [%1];" : "=r"( x ) : _ASM_PTR_(addr));
    return x;
#   else
    return *addr;
#   endif
}

template< typename T >
__device__ __forceinline__ void storeGlobal32( T* addr, T const & val )
{
#   if (__CUDA_ARCH__ < 700)
    asm volatile( "st.global.cg.u32 [%0], %1;" : : _ASM_PTR_(addr), "r"( val ) );
#   else
    *addr = val;
#   endif
}

template< typename T >
__device__ __forceinline__ void storeGlobal64( T* addr, T const & val )
{
#   if (__CUDA_ARCH__ < 700)
    asm volatile("st.global.cg.u64 [%0], %1;" : : _ASM_PTR_(addr), "l"(val));
#   else
    *addr = val;
#   endif
}

template<size_t ITERATIONS, uint32_t MEM>
__global__ void cryptonight_core_gpu_phase1( int threads, int bfactor, int partidx, uint32_t * __restrict__ long_state, uint32_t * __restrict__ ctx_state2, uint32_t * __restrict__ ctx_key1 )
{
    __shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init( sharedMemory );
    __syncthreads( );

    const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
    const int sub = ( threadIdx.x & 7 ) << 2;

    const int batchsize = MEM >> bfactor;
    const int start = partidx * batchsize;
    const int end = start + batchsize;

    if ( thread >= threads )
        return;

    uint32_t key[40], text[4];

    MEMCPY8( key, ctx_key1 + thread * 40, 20 );

    if (partidx == 0) {
        // first round
        MEMCPY8( text, ctx_state2 + thread * 50 + sub + 16, 2 );
    }
    else {
        // load previous text data
        MEMCPY8( text, &long_state[( (uint64_t) thread * MEM) + sub + start - 32], 2 );
    }

    __syncthreads( );
    for (int i = start; i < end; i += 32) {
        cn_aes_pseudo_round_mut( sharedMemory, text, key );
        MEMCPY8(&long_state[((uint64_t) thread * MEM) + (sub + i)], text, 2);
    }
}

/** avoid warning `unused parameter` */
template< typename T >
__forceinline__ __device__ void unusedVar( const T& )
{
}

/** shuffle data for
 *
 * - this method can be used with all compute architectures
 * - for <sm_30 shared memory is needed
 *
 * group_n - must be a power of 2!
 *
 * @param ptr pointer to shared memory, size must be `threadIdx.x * sizeof(uint32_t)`
 *            value can be NULL for compute architecture >=sm_30
 * @param sub thread number within the group, range [0:group_n]
 * @param value value to share with other threads within the group
 * @param src thread number within the group from where the data is read, range [0:group_n]
 */
template<size_t group_n>
__forceinline__ __device__ uint32_t shuffle(volatile uint32_t* ptr,const uint32_t sub,const int val,const uint32_t src)
{
#   if ( __CUDA_ARCH__ < 300 )
    ptr[sub] = val;
    return ptr[src & (group_n-1)];
#   else
    unusedVar( ptr );
    unusedVar( sub );
#   if (__CUDACC_VER_MAJOR__ >= 9)
    return __shfl_sync(__activemask(), val, src, group_n);
#   else
    return __shfl( val, src, group_n );
#   endif
#   endif
}


template<size_t group_n>
__forceinline__ __device__ uint64_t shuffle64(volatile uint32_t* ptr,const uint32_t sub,const int val,const uint32_t src, const uint32_t src2)
{
    uint64_t tmp;
    ((uint32_t*)&tmp)[0] = shuffle<group_n>(ptr, sub, val, src);
    ((uint32_t*)&tmp)[1] = shuffle<group_n>(ptr, sub, val, src2);
    return tmp;
}

struct u64 : public uint2
{

    __forceinline__ __device__ u64(){}

    __forceinline__ __device__ u64( const uint32_t x0, const uint32_t x1)
    {
        uint2::x = x0;
        uint2::y = x1;
    }

    __forceinline__ __device__ operator uint64_t() const
    {
        return *((uint64_t*)this);
    }

    __forceinline__ __device__ u64( const uint64_t x0)
    {
        ((uint64_t*)&this->x)[0] = x0;
    }

    __forceinline__ __device__ u64 operator^=(const u64& other)
    {
        uint2::x ^= other.x;
        uint2::y ^= other.y;

        return *this;
    }

    __forceinline__ __device__ u64 operator+(const u64& other) const
    {
        u64 tmp;
        ((uint64_t*)&tmp.x)[0] = ((uint64_t*)&(this->x))[0] + ((uint64_t*)&(other.x))[0];

        return tmp;
    }

    __forceinline__ __device__ u64 operator+=(const uint64_t& other)
    {
        return ((uint64_t*)&this->x)[0] += other;
    }
};

/** cryptonight with two threads per hash
 */
template<size_t ITERATIONS, uint32_t MEM, uint32_t MASK, xmrig::Algorithm::Id ALGO>
#ifdef XMRIG_THREADS
__launch_bounds__( XMRIG_THREADS * 2 )
#endif
__global__ void cryptonight_core_gpu_phase2_double(
        int threads,
        int bfactor,
        int partidx,
        uint32_t *d_long_state,
        uint32_t *d_ctx_a,
        uint32_t *d_ctx_b,
        uint32_t * d_ctx_state,
        uint32_t startNonce,
        uint32_t * __restrict__ d_input
        )
{
    using namespace xmrig;

    __shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init( sharedMemory );

#   if( __CUDA_ARCH__ < 300 )
    extern __shared__ uint64_t externShared[];
    // 8 x 64bit values
    volatile uint64_t* myChunks = (volatile uint64_t*)(externShared + (threadIdx.x >> 1) * 8);
    volatile uint32_t* sPtr = (volatile uint32_t*)(externShared + (blockDim.x >> 1) * 8)  + (threadIdx.x & 0xFFFFFFFE);
#   else
    extern __shared__ uint64_t chunkMem[];
    volatile uint32_t* sPtr = NULL;
    // 8 x 64bit values
    volatile uint64_t* myChunks = (volatile uint64_t*)(chunkMem + (threadIdx.x >> 1) * 8);
#   endif

    __syncthreads( );

    const uint64_t tid    = (blockDim.x * blockIdx.x + threadIdx.x);
    const uint32_t thread = tid >> 1;
    const uint32_t sub    = tid & 1;

    if (thread >= threads) {
        return;
    }

    uint8_t *l0              = (uint8_t*)&d_long_state[(IndexType) thread * MEM];
    uint64_t ax0             = ((uint64_t*)(d_ctx_a + thread * 4))[sub];
    uint32_t idx0            = shuffle<2>(sPtr, sub, static_cast<uint32_t>(ax0), 0);
    uint64_t bx0             = ((uint64_t*)(d_ctx_b + thread * 16))[sub];
    uint64_t bx1             = ((uint64_t*)(d_ctx_b + thread * 16 + 4))[sub];
    uint64_t division_result = ((uint64_t*)(d_ctx_b + thread * 16 + 4 * 2))[0];
    uint32_t sqrt_result     = (d_ctx_b + thread * 16 + 4 * 2 + 2)[0];

    const int batchsize      = (ITERATIONS * 2) >> ( 1 + bfactor );
    const int start          = partidx * batchsize;
    const int end            = start + batchsize;

    uint64_t* ptr0;
    for (int i = start; i < end; ++i) {
        ptr0 = (uint64_t *)&l0[idx0 & (MASK - 0x30)];

        ((ulonglong4*)myChunks)[sub] = ((ulonglong4*)ptr0)[sub];

        uint32_t idx1 = (idx0 & 0x30) >> 3;
        const u64 cx  = myChunks[ idx1 + sub ];
        const u64 cx2 = myChunks[ idx1 + ((sub + 1) & 1) ];

        u64 cx_aes = ax0 ^ u64(
            t_fn0( cx.x & 0xff ) ^ t_fn1( (cx.y >> 8) & 0xff ) ^ t_fn2( (cx2.x >> 16) & 0xff ) ^ t_fn3( (cx2.y >> 24 ) ),
            t_fn0( cx.y & 0xff ) ^ t_fn1( (cx2.x >> 8) & 0xff ) ^ t_fn2( (cx2.y >> 16) & 0xff ) ^ t_fn3( (cx.x >> 24 ) )
        );

        {
            const uint64_t chunk1 = myChunks[idx1 ^ 2 + sub];
            const uint64_t chunk2 = myChunks[idx1 ^ 4 + sub];
            const uint64_t chunk3 = myChunks[idx1 ^ 6 + sub];

#           if (__CUDACC_VER_MAJOR__ >= 9)
            __syncwarp();
#           else
            __syncthreads();
#           endif

            myChunks[idx1 ^ 2 + sub] = ((ALGO == Algorithm::CN_RWZ) ? chunk1 : chunk3) + bx1;
            myChunks[idx1 ^ 4 + sub] = ((ALGO == Algorithm::CN_RWZ) ? chunk3 : chunk1) + bx0;
            myChunks[idx1 ^ 6 + sub] = chunk2 + ax0;
        }

        myChunks[idx1 + sub] = cx_aes ^ bx0;

        ((ulonglong4*)ptr0)[sub] = ((ulonglong4*)myChunks)[sub];

        idx0 = shuffle<2>(sPtr, sub, cx_aes.x, 0);
        idx1 = (idx0 & 0x30) >> 3;
        ptr0 = (uint64_t *)&l0[idx0 & MASK & (MASK - 0x30)];

        ((ulonglong4*)myChunks)[sub] = ((ulonglong4*)ptr0)[sub];

        uint64_t cx_mul;
        ((uint32_t*)&cx_mul)[0] = shuffle<2>(sPtr, sub, cx_aes.x , 0);
        ((uint32_t*)&cx_mul)[1] = shuffle<2>(sPtr, sub, cx_aes.y , 0);

        if (sub == 1) {
            // Use division and square root results from the _previous_ iteration to hide the latency
            ((uint32_t*)&division_result)[1] ^= sqrt_result;
            ((uint64_t*)myChunks)[idx1]      ^= division_result;

            const uint32_t dd = (static_cast<uint32_t>(cx_mul) + (sqrt_result << 1)) | 0x80000001UL;
            division_result = fast_div_v2(cx_aes, dd);

            // Use division_result as an input for the square root to prevent parallel implementation in hardware
            sqrt_result = fast_sqrt_v2(cx_mul + division_result);
        }

#       if (__CUDACC_VER_MAJOR__ >= 9)
        __syncwarp();
#       else
        __syncthreads( );
#       endif

        uint64_t c = ((uint64_t*)myChunks)[idx1 + sub];

        {
            uint64_t cl = ((uint64_t*)myChunks)[idx1];
            // sub 0 -> hi, sub 1 -> lo
            uint64_t res = sub == 0 ? __umul64hi( cx_mul, cl ) : cx_mul * cl;

            const uint64_t chunk1 = myChunks[ idx1 ^ 2 + sub ] ^ res;
            uint64_t chunk2       = myChunks[ idx1 ^ 4 + sub ];
            res ^= ((uint64_t*)&chunk2)[0];
            const uint64_t chunk3 = myChunks[ idx1 ^ 6 + sub ];

#           if (__CUDACC_VER_MAJOR__ >= 9)
            __syncwarp();
#           else
            __syncthreads( );
#           endif

            myChunks[idx1 ^ 2 + sub] = ((ALGO == Algorithm::CN_RWZ) ? chunk1 : chunk3) + bx1;
            myChunks[idx1 ^ 4 + sub] = ((ALGO == Algorithm::CN_RWZ) ? chunk3 : chunk1) + bx0;
            myChunks[idx1 ^ 6 + sub] = chunk2 + ax0;

            ax0 += res;
        }

        bx1 = bx0;
        bx0 = cx_aes;

        myChunks[idx1 + sub] = ax0;

        ((ulonglong4*)ptr0)[sub] = ((ulonglong4*)myChunks)[sub];

        ax0 ^= c;
        idx0 = shuffle<2>(sPtr, sub, static_cast<uint32_t>(ax0), 0);
    }

    if (bfactor > 0) {
        ((uint64_t*)(d_ctx_a + thread * 4))[sub]      = ax0;
        ((uint64_t*)(d_ctx_b + thread * 16))[sub]     = bx0;
        ((uint64_t*)(d_ctx_b + thread * 16 + 4))[sub] = bx1;

        if (sub == 1) {
            // must be valid only for `sub == 1`
            ((uint64_t*)(d_ctx_b + thread * 16 + 4 * 2))[0] = division_result;
            (d_ctx_b + thread * 16 + 4 * 2 + 2)[0]          = sqrt_result;
        }
    }
}


template<size_t ITERATIONS, uint32_t MEM, uint32_t MASK, xmrig::Algorithm::Id ALGO, xmrig::Algorithm::Id BASE>
#ifdef XMRIG_THREADS
__launch_bounds__( XMRIG_THREADS * 4 )
#endif
__global__ void cryptonight_core_gpu_phase2_quad(
        int threads,
        int bfactor,
        int partidx,
        uint32_t *d_long_state,
        uint32_t *d_ctx_a,
        uint32_t *d_ctx_b,
        uint32_t *d_ctx_state,
        uint32_t startNonce,
        uint32_t *__restrict__ d_input
        )
{
    using namespace xmrig;

    __shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init( sharedMemory );

    __syncthreads( );

    const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 2;
    const uint32_t nonce = startNonce + thread;
    const int sub = threadIdx.x & 3;
    const int sub2 = sub & 2;

#if( __CUDA_ARCH__ < 300 )
        extern __shared__ uint32_t shuffleMem[];
        volatile uint32_t* sPtr = (volatile uint32_t*)(shuffleMem + (threadIdx.x& 0xFFFFFFFC));
#else
        volatile uint32_t* sPtr = NULL;
#endif
    if (thread >= threads) {
        return;
    }

    int i, k;
    uint32_t j;
    const int batchsize = (ITERATIONS * 2) >> (2 + bfactor);
    const int start = partidx * batchsize;
    const int end = start + batchsize;
    uint32_t * long_state = &d_long_state[(IndexType) thread * MEM];
    uint32_t a, d[2], idx0;
    uint32_t t1[2], t2[2], res;

    uint32_t tweak1_2[2];
    if (BASE == Algorithm::CN_1) {
        uint32_t * state = d_ctx_state + thread * 50;
        tweak1_2[0] = (d_input[8] >> 24) | (d_input[9] << 8);
        tweak1_2[0] ^= state[48];
        tweak1_2[1] = nonce;
        tweak1_2[1] ^= state[49];
    }

    a = (d_ctx_a + thread * 4)[sub];
    idx0 = shuffle<4>(sPtr,sub, a, 0);
    if (ALGO == Algorithm::CN_HEAVY_0 || ALGO == Algorithm::CN_HEAVY_TUBE || ALGO == Algorithm::CN_HEAVY_XHV) {
        if (partidx != 0) {
            // state is stored after all ctx_b states
            idx0 = *(d_ctx_b + threads * 4 + thread);
        }
    }

    d[1] = (d_ctx_b + thread * 4)[sub];

    float conc_var;
    if (ALGO == Algorithm::CN_CCX || ALGO == Algorithm::CN_CACHE_HASH) {
        conc_var = (partidx != 0) ? int_as_float(*(d_ctx_b + threads * 4 + thread * 4 + sub)) : 0.0f;
    }

    #pragma unroll 2
    for (i = start; i < end; ++i) {
        #pragma unroll 2
        for (int x = 0; x < 2; ++x) {
            j = ((idx0 & MASK) >> 2) + sub;

            if (ALGO == Algorithm::CN_HEAVY_TUBE) {
                uint32_t k[4];
                k[0] = ~loadGlobal32<uint32_t>(long_state + j);
                k[1] = shuffle<4>(sPtr,sub, k[0], sub + 1);
                k[2] = shuffle<4>(sPtr,sub, k[0], sub + 2);
                k[3] = shuffle<4>(sPtr,sub, k[0], sub + 3);

                #pragma unroll 4
                for (int i = 0; i < 4; ++i) {
                    // only calculate the key if all data are up to date
                    if (i == sub) {
                        d[x] = a ^
                            t_fn0(k[0] & 0xff) ^
                            t_fn1((k[1] >> 8) & 0xff) ^
                            t_fn2((k[2] >> 16) & 0xff) ^
                            t_fn3((k[3] >> 24));
                    }
                    // the last shuffle is not needed
                    if (i != 3) {
                        /* avoid negative number for modulo
                         * load valid key (k) depending on the round
                         */
                        k[(4 - sub + i) % 4] = shuffle<4>(sPtr,sub, k[0] ^ d[x], i);
                    }
                }
            } else {
                uint32_t x_0 = loadGlobal32<uint32_t>(long_state + j);

                if (ALGO == Algorithm::CN_CCX || ALGO == Algorithm::CN_CACHE_HASH) {
                    float r = int2float((int32_t)x_0) + conc_var;
                    r = int_as_float((float_as_int(r * r * r) & 0x807FFFFF) | 0x40000000);
                    x_0 ^= (int32_t)(int_as_float((float_as_int(conc_var) & 0x807FFFFF) | 0x40000000) * 536870880.0f);
                    conc_var += r;
                }

                const uint32_t x_1 = shuffle<4>(sPtr,sub, x_0, sub + 1);
                const uint32_t x_2 = shuffle<4>(sPtr,sub, x_0, sub + 2);
                const uint32_t x_3 = shuffle<4>(sPtr,sub, x_0, sub + 3);
                d[x] = a ^
                    t_fn0(x_0 & 0xff) ^
                    t_fn1((x_1 >> 8) & 0xff) ^
                    t_fn2((x_2 >> 16) & 0xff) ^
                    t_fn3((x_3 >> 24));
            }

            //XOR_BLOCKS_DST(c, b, &long_state[j]);
            t1[0] = shuffle<4>(sPtr,sub, d[x], 0);

            const uint32_t z = d[0] ^ d[1];
            if (BASE == Algorithm::CN_1) {
                const uint32_t table = 0x75310U;
                const uint32_t index = ((z >> (26)) & 12) | ((z >> 23) & 2);
                const uint32_t fork_7 = z ^ ((table >> index) & 0x30U) << 24;
                storeGlobal32(long_state + j, sub == 2 ? fork_7 : z);
            }
            else {
                storeGlobal32(long_state + j, z);
            }

            //MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & MASK]);
            j = ( ( *t1 & MASK ) >> 2 ) + sub;

            uint32_t yy[2];
            *( (uint64_t*) yy ) = loadGlobal64<uint64_t>( ( (uint64_t *) long_state )+( j >> 1 ) );
            uint32_t zz[2];
            zz[0] = shuffle<4>(sPtr,sub, yy[0], 0);
            zz[1] = shuffle<4>(sPtr,sub, yy[1], 0);

            t1[1] = shuffle<4>(sPtr,sub, d[x], 1);
            #pragma unroll
            for (k = 0; k < 2; k++) {
                t2[k] = shuffle<4>(sPtr,sub, a, k + sub2);
            }

            *( (uint64_t *) t2 ) += sub2 ? ( *( (uint64_t *) t1 ) * *( (uint64_t*) zz ) ) : __umul64hi( *( (uint64_t *) t1 ), *( (uint64_t*) zz ) );

            res = *( (uint64_t *) t2 )  >> ( sub & 1 ? 32 : 0 );

            if (BASE == Algorithm::CN_1) {
                const uint32_t tweaked_res = tweak1_2[sub & 1] ^ res;
                uint32_t long_state_update = sub2 ? tweaked_res : res;

                if (ALGO == Algorithm::CN_HEAVY_TUBE || ALGO == Algorithm::CN_RTO) {
                    uint32_t value = shuffle<4>(sPtr,sub, long_state_update, sub & 1) ^ long_state_update;
                    long_state_update = sub >= 2 ? value : long_state_update;
                }

                storeGlobal32(long_state + j, long_state_update);
            }
            else {
                storeGlobal32(long_state + j, res);
            }

            a = ( sub & 1 ? yy[1] : yy[0] ) ^ res;
            idx0 = shuffle<4>(sPtr,sub, a, 0);
            if (ALGO == Algorithm::CN_HEAVY_0 || ALGO == Algorithm::CN_HEAVY_TUBE || ALGO == Algorithm::CN_HEAVY_XHV) {
                int64_t n = loadGlobal64<uint64_t>( ( (uint64_t *) long_state ) + (( idx0 & MASK ) >> 3));
                int32_t d = loadGlobal32<uint32_t>( (uint32_t*)(( (uint64_t *) long_state ) + (( idx0 & MASK) >> 3) + 1u ));
                int64_t q = fast_div_heavy(n, d | 0x5);

                if (sub & 1) {
                    storeGlobal64<uint64_t>( ( (uint64_t *) long_state ) + (( idx0 & MASK ) >> 3), n ^ q );
                }

                if (ALGO == Algorithm::CN_HEAVY_XHV) {
                    d = ~d;
                }

                idx0 = d ^ q;
            }
        }
    }

    if (bfactor > 0) {
        (d_ctx_a + thread * 4)[sub] = a;
        (d_ctx_b + thread * 4)[sub] = d[1];
        if (ALGO == Algorithm::CN_HEAVY_0 || ALGO == Algorithm::CN_HEAVY_TUBE || ALGO == Algorithm::CN_HEAVY_XHV) {
            if (sub&1) {
                *(d_ctx_b + threads * 4 + thread) = idx0;
            }
        }
        if (ALGO == Algorithm::CN_CCX || ALGO == Algorithm::CN_CACHE_HASH) {
            *(d_ctx_b + threads * 4 + thread * 4 + sub) = float_as_int(conc_var);
        }
    }
}

template<size_t ITERATIONS, uint32_t MEM, xmrig::Algorithm::Id ALGO>
__global__ void cryptonight_core_gpu_phase3( int threads, int bfactor, int partidx, const uint32_t * __restrict__ long_state, uint32_t * __restrict__ d_ctx_state, uint32_t * __restrict__ d_ctx_key2 )
{
    using namespace xmrig;

    __shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init( sharedMemory );
    __syncthreads( );

    int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
    int subv = ( threadIdx.x & 7 );
    int sub = subv << 2;

    const int batchsize = MEM >> bfactor;
    const int start = (partidx % (1 << bfactor)) * batchsize;
    const int end = start + batchsize;

    if ( thread >= threads )
        return;

    uint32_t key[40], text[4];
    MEMCPY8( key, d_ctx_key2 + thread * 40, 20 );
    MEMCPY8( text, d_ctx_state + thread * 50 + sub + 16, 2 );

    __syncthreads( );

#   if ( __CUDA_ARCH__ < 300 )
    extern __shared__ uint32_t shuffleMem[];
    volatile uint32_t* sPtr = (volatile uint32_t*)(shuffleMem + (threadIdx.x& 0xFFFFFFF8));
#   else
    volatile uint32_t* sPtr = NULL;
#   endif

    for (int i = start; i < end; i += 32) {
#pragma unroll
        for (int j = 0; j < 4; ++j) {
            text[j] ^= long_state[((IndexType) thread * MEM) + ( sub + i + j)];
        }

        cn_aes_pseudo_round_mut( sharedMemory, text, key );

        if (ALGO == Algorithm::CN_HEAVY_0 || ALGO == Algorithm::CN_HEAVY_TUBE || ALGO == Algorithm::CN_HEAVY_XHV) {
            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                text[j] ^= shuffle<8>(sPtr, subv, text[j], (subv+1) & 7);
            }
        }
    }

    MEMCPY8( d_ctx_state + thread * 50 + sub + 16, text, 2 );
}

template<xmrig::Algorithm::Id ALGO>
void cryptonight_core_gpu_hash(nvid_ctx* ctx, uint32_t nonce)
{
    using namespace xmrig;
    constexpr CnAlgo<ALGO> props;

    constexpr size_t MASK         = props.mask();
    constexpr size_t ITERATIONS   = props.iterations();
    constexpr size_t MEM          = props.memory() / 4;
    constexpr Algorithm::Id BASE  = props.base();

//    printf("%x\n", MASK);
//    printf("%x\n", ITERATIONS);
//    printf("%x %x\n", MEM, props.memory());
//    printf("%x\n", BASE);

    dim3 grid(ctx->device_blocks);
    dim3 block(ctx->device_threads);
    dim3 block2(ctx->device_threads << 1);
    dim3 block4(ctx->device_threads << 2);
    dim3 block8(ctx->device_threads << 3);

    int partcount = 1 << ctx->device_bfactor;

    /* bfactor for phase 1 and 3
     *
     * phase 1 and 3 consume less time than phase 2, therefore we begin with the
     * kernel splitting if the user defined a `bfactor >= 5`
     */
    int bfactorOneThree = ctx->device_bfactor - 4;
    if (bfactorOneThree < 0) {
        bfactorOneThree = 0;
    }

    const int partcountOneThree = 1 << bfactorOneThree;
    for (int i = 0; i < partcountOneThree; i++) {
        CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase1<ITERATIONS, MEM><<< grid, block8 >>>( ctx->device_blocks*ctx->device_threads,
            bfactorOneThree, i,
            ctx->d_long_state,
            (props.isHeavy() ? ctx->d_ctx_state2 : ctx->d_ctx_state),
            ctx->d_ctx_key1));

        if (partcount > 1 && ctx->device_bsleep > 0) {
            compat_usleep(ctx->device_bsleep);
        }
    }

    if (partcount > 1 && ctx->device_bsleep > 0) {
        compat_usleep(ctx->device_bsleep);
    }

    for (int i = 0; i < partcount; i++) {
#       ifdef XMRIG_DRIVER_API
        if (ALGO == Algorithm::CN_R) {
            int threads = ctx->device_blocks * ctx->device_threads;
            void* args[] = { &threads, &ctx->device_bfactor, &i, &ctx->d_long_state, &ctx->d_ctx_a, &ctx->d_ctx_b, &ctx->d_ctx_state, &nonce, &ctx->d_input };
            CU_CHECK(ctx->device_id, hipModuleLaunchKernel(
                ctx->kernel,
                grid.x, grid.y, grid.z,
                block2.x, block2.y, block2.z,
                sizeof(uint64_t) * block.x * 8 + block.x * sizeof(uint32_t) * static_cast<int>(ctx->device_arch[0] < 3), nullptr,
                args, 0
            ));
            CU_CHECK(ctx->device_id, hipCtxSynchronize());
        } else
#       endif
        if (BASE == Algorithm::CN_2) {
            CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase2_double<ITERATIONS, MEM, MASK, ALGO><<<
                grid,
                block2,
                sizeof(uint64_t) * block.x * 8 + block.x * sizeof(uint32_t) * static_cast<int>(ctx->device_arch[0] < 3)
            >>>(
                ctx->device_blocks * ctx->device_threads,
                ctx->device_bfactor,
                i,
                ctx->d_long_state,
                ctx->d_ctx_a,
                ctx->d_ctx_b,
                ctx->d_ctx_state,
                nonce,
                ctx->d_input
                )
            );
        } else {
            CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase2_quad<ITERATIONS, MEM, MASK, ALGO, BASE><<<
                grid,
                block4,
                block4.x * sizeof(uint32_t) * static_cast<int>(ctx->device_arch[0] < 3)
            >>>(
                ctx->device_blocks * ctx->device_threads,
                ctx->device_bfactor,
                i,
                ctx->d_long_state,
                ctx->d_ctx_a,
                ctx->d_ctx_b,
                ctx->d_ctx_state,
                nonce,
                ctx->d_input
                )
            );
        }

        if (partcount > 1 && ctx->device_bsleep > 0) {
            compat_usleep(ctx->device_bsleep);
        }
    }

    const int roundsPhase3 = props.isHeavy() ? partcountOneThree * 2 : partcountOneThree;
    for (int i = 0; i < roundsPhase3; i++) {
        CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase3<ITERATIONS, MEM, ALGO><<<
            grid,
            block8,
            block8.x * sizeof(uint32_t) * static_cast< int >( ctx->device_arch[0] < 3 )
        >>>( ctx->device_blocks*ctx->device_threads,
            bfactorOneThree, i,
            ctx->d_long_state,
            ctx->d_ctx_state, ctx->d_ctx_key2));
    }
}


void cryptonight_gpu_hash(nvid_ctx *ctx, const xmrig::Algorithm &algorithm, uint64_t height, uint32_t startNonce)
{
    using namespace xmrig;

    if (algorithm.family() == Algorithm::CN) {
        if (algorithm == Algorithm::CN_R) {
#           ifdef XMRIG_DRIVER_API
            if ((ctx->algorithm != algorithm) || (ctx->kernel_height != height)) {
                if (ctx->module) {
                    hipModuleUnload(ctx->module);
                }

                std::vector<char> ptx;
                std::string lowered_name;
                CryptonightR_get_program(ptx, lowered_name, height, ctx->device_arch[0], ctx->device_arch[1]); // FIXME

                CU_CHECK(ctx->device_id, hipModuleLoadDataEx(&ctx->module, ptx.data(), 0, 0, 0));
                CU_CHECK(ctx->device_id, hipModuleGetFunction(&ctx->kernel, ctx->module, lowered_name.c_str()));

                ctx->algorithm      = algorithm;
                ctx->kernel_height  = height;

                CryptonightR_get_program(ptx, lowered_name, height + 1, ctx->device_arch[0], ctx->device_arch[1], true); // FIXME
            }
#           endif
        }

        switch (algorithm.id()) {
        case Algorithm::CN_0:
            cryptonight_core_gpu_hash<Algorithm::CN_0>(ctx, startNonce);
            break;

        case Algorithm::CN_1:
            cryptonight_core_gpu_hash<Algorithm::CN_1>(ctx, startNonce);
            break;

        case Algorithm::CN_2:
            cryptonight_core_gpu_hash<Algorithm::CN_2>(ctx, startNonce);
            break;

#       ifdef XMRIG_DRIVER_API
        case Algorithm::CN_R:
            cryptonight_core_gpu_hash<Algorithm::CN_R>(ctx, startNonce);
            break;
#       endif

        case Algorithm::CN_FAST:
            cryptonight_core_gpu_hash<Algorithm::CN_FAST>(ctx, startNonce);
            break;

        case Algorithm::CN_HALF:
            cryptonight_core_gpu_hash<Algorithm::CN_HALF>(ctx, startNonce);
            break;

        case Algorithm::CN_XAO:
            cryptonight_core_gpu_hash<Algorithm::CN_XAO>(ctx, startNonce);
            break;

        case Algorithm::CN_RTO:
            cryptonight_core_gpu_hash<Algorithm::CN_RTO>(ctx, startNonce);
            break;

        case Algorithm::CN_RWZ:
            cryptonight_core_gpu_hash<Algorithm::CN_RWZ>(ctx, startNonce);
            break;

        case Algorithm::CN_ZLS:
            cryptonight_core_gpu_hash<Algorithm::CN_ZLS>(ctx, startNonce);
            break;

        case Algorithm::CN_DOUBLE:
            cryptonight_core_gpu_hash<Algorithm::CN_DOUBLE>(ctx, startNonce);
            break;

        case Algorithm::CN_CCX:
            cryptonight_core_gpu_hash<Algorithm::CN_CCX>(ctx, startNonce);
            break;

        case Algorithm::CN_CACHE_HASH:
            cryptonight_core_gpu_hash<Algorithm::CN_CACHE_HASH>(ctx, startNonce);
            break;

        default:
            break;
        }
    }
    else if (algorithm.family() == Algorithm::CN_LITE) {
        switch (algorithm.id()) {
        case Algorithm::CN_LITE_0:
            cryptonight_core_gpu_hash<Algorithm::CN_LITE_0>(ctx, startNonce);
            break;

        case Algorithm::CN_LITE_1:
            cryptonight_core_gpu_hash<Algorithm::CN_LITE_1>(ctx, startNonce);
            break;

        default:
            break;
        }
    }
    else if (algorithm.family() == Algorithm::CN_HEAVY) {
        switch (algorithm.id()) {
        case Algorithm::CN_HEAVY_0:
            cryptonight_core_gpu_hash<Algorithm::CN_HEAVY_0>(ctx, startNonce);
            break;

        case Algorithm::CN_HEAVY_TUBE:
            cryptonight_core_gpu_hash<Algorithm::CN_HEAVY_TUBE>(ctx, startNonce);
            break;

        case Algorithm::CN_HEAVY_XHV:
            cryptonight_core_gpu_hash<Algorithm::CN_HEAVY_XHV>(ctx, startNonce);
            break;

        default:
            break;
        }
    }
    else if (algorithm.family() == Algorithm::CN_PICO) {
        switch (algorithm.id()) {
        case Algorithm::CN_PICO_0:
            cryptonight_core_gpu_hash<Algorithm::CN_PICO_0>(ctx, startNonce);
            break;

        case Algorithm::CN_PICO_TLO:
            cryptonight_core_gpu_hash<Algorithm::CN_PICO_TLO>(ctx, startNonce);
            break;

        default:
            break;
        }
    }
}
